#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void reduce_first_add(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
    sdata[tid] = g_idata[i] + g_idata[i+blockDim.x];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if (tid % (2*s) == 0) {  // will cause divergence, for the result of "if" is different
            sdata[tid] += sdata[tid + s];
        }
        //printf("res: %d\n", sdata[tid]);
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    }
}