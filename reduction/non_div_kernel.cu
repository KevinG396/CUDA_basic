#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void reduce_non_div(int *g_idata, int *g_odata) {
    extern __shared__ int sdata[];
    // each thread loads one element from global to shared mem
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    sdata[tid] = g_idata[i];
    __syncthreads();
    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        int index = 2 * s * tid;
        if (index < blockDim.x) {    // same result, non-divergence
        sdata[index] += sdata[index + s];
        }
        //printf("res: %d\n", sdata[tid]);
        __syncthreads();
    }
    // write result for this block to global mem
    if (tid == 0){
        g_odata[blockIdx.x] = sdata[0];
    }
}
