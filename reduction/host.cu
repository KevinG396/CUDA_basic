#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernel.h"

int main() {
    int size = 1024;
    int bytes = size * sizeof(int); // array mem use
    int threadsPerBlock = 256;
    int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    int *h_idata = (int*) malloc(bytes); // inp host
    int *h_odata = (int*) malloc(blocksPerGrid*sizeof(int)); // oup host
    int *d_idata, *d_odata;
    int sum = 0;
    // initialize
    for(int i = 0; i < size; i++) {
        h_idata[i] = 1;
    }
    // allocate mem on device
    hipMalloc((void**)&d_idata, bytes);
    hipMalloc((void**)&d_odata, blocksPerGrid*sizeof(int));

    // cp from host 2 device
    hipMemcpy(d_idata, h_idata, bytes, hipMemcpyHostToDevice);

    // execute kernel
    //int threadsPerBlock = 256; // 256 threads per block
    //int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
    reduce0<<<blocksPerGrid, threadsPerBlock>>>(d_idata, d_odata);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
    }
    //  cp result from device 2 host
    hipMemcpy(h_odata, d_odata, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

    // finish reduction

    for(int i = 0; i < blocksPerGrid; i++) {
        printf("sum: %d\n", sum);
        sum += h_odata[i];
    }

    printf("Total Sum = %d\n", sum);

    // free
    hipFree(d_idata);
    hipFree(d_odata);
    free(h_idata);
    free(h_odata);

    return 0;
}